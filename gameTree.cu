
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdint.h>
#include <stdio.h>
#include <time.h>


/*
enum Piece
{
	empty,
	white_reg,
	white_reg_moved,
	white_king,
	white_king_moved,
	black_reg,
	black_reg_moved,
	black_king,
	black_king_moved
};*/

typedef uint8_t Piece;
const Piece empty = 0;
const Piece white_reg = empty + 1;
const Piece white_reg_moved = white_reg + 1;
const Piece white_king = white_reg_moved + 1;
const Piece white_king_moved = white_king + 1;
const Piece black_reg = white_king_moved + 1;
const Piece black_reg_moved = black_reg + 1;
const Piece black_king = black_reg_moved + 1;
const Piece black_king_moved = black_king + 1;

struct Board {
	Piece pieces[4][8];
};
enum Turn
{
	white,
	black
};
struct Pair {
	unsigned char first;
	unsigned char second;
};

__constant__ Board bad_board = {empty};
Board bad_board_host = {empty};

#define USE_GPU 1
#if USE_GPU

#define BLOCK_SIZE 512
#define gpuErrChk(stmt) \
do\
{\
	hipError_t errCode = stmt; \
	if(errCode != hipSuccess)\
	{ \
		std::cerr << "gpuErrChk: " << hipGetErrorString(errCode)\
			<< " " <<  __FILE__ << " " <<  __LINE__ << " "\
			<< std::endl;\
		return -1;\
	}\
} while(0)

__device__ Board outputBoard;
__device__ void makeMoves(Board * boards, Turn turn, unsigned int tx);

__device__ bool boardEquality(const Board *a, const Board*b)
{
	for(int x = 0; x < 4; x++)
	{
		for(int y = 0; y < 8; y++)
		{
			if(a->pieces[x][y] != b->pieces[x][y])
			{
				return false;
			}
		}
	}
	return true;
}
bool boardEqualityHost(const Board *a, const Board*b)
{
	for(int x = 0; x < 4; x++)
	{
		for(int y = 0; y < 8; y++)
		{
			if(a->pieces[x][y] != b->pieces[x][y])
			{
				return false;
			}
		}
	}
	return true;
}
__global__ void analyze_tree(Board * input, int moves){
	int max = 0;
}

__global__ void expand(Board * input, Board * output, int len) {
	const int shared_size = 496;
	__shared__ Board B[shared_size];
	unsigned int tx = threadIdx.x;
	unsigned int blockNum = blockIdx.x+blockIdx.y*gridDim.x;
	
	if (blockNum < len && tx == 0)
	{
		B[0] = input[blockNum];
	}
	else if (blockNum < len && tx < shared_size)
	{
		B[tx] = bad_board;
	}	
	__syncthreads();
	if(tx == 0 && ~boardEquality(&B[tx], &bad_board))
		makeMoves(B, white, tx);
	__syncthreads();
	if(tx < shared_size && ~boardEquality(&B[tx], &bad_board))
		makeMoves(B, black, tx);
	__syncthreads();

	if (tx < shared_size && blockNum < len)
		output[blockDim.x*blockNum+tx] = B[tx];
	else if (blockNum < len)
		output[blockDim.x*blockNum+tx] = bad_board;
}


//TODO: deal with 22 move boundary
__device__ 
#endif
void makeMoves(Board * boards, Turn turn, unsigned int tx)
{
	// tx = 0 condition because only the first thread has a valid board to work on.
	if(turn == white && tx == 0)
	{
		int exp_rate = 22;
		int move_idx = 0;
		Board b = boards[tx];
		Board temp = boards[tx];
		for(int x = 0; x < 4; x++)
		for(int y = 0; y < 8; y++)
		{
			if (b.pieces[x][y] == white_reg || b.pieces[x][y] == white_king)
			{
				/*White pieces move (not take) */
				if(y%2 && y < 6 && x != 3 && !b.pieces[x+1][y+1]) 
				{	
					//printf("white at %d,%d move right\n", x, y);
					temp.pieces[x+1][y+1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(y%2 && y < 6 && !b.pieces[x][y+1])
				{
					//printf("white at %d,%d move left\n", x, y);
					temp.pieces[x][y+1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && x != 0 && !b.pieces[x-1][y+1])
				{
					//printf("white at %d,%d move left\n", x, y);
					if (y == 6)
						temp.pieces[x-1][y+1] = white_king;
					else
						temp.pieces[x-1][y+1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && !b.pieces[x][y+1])
				{
					//printf("white at %d,%d move right\n", x, y);
					if (y == 6)
						temp.pieces[x][y+1] = white_king;
					else
						temp.pieces[x][y+1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				/*White piece captures a black piece (not become king)*/
				if(y%2 && x!= 3 && b.pieces[x+1][y+1] > white_king_moved && !b.pieces[x+1][y+2]) 
				{
					//TODO add double takes here
					if (y != 5)
						temp.pieces[x+1][y+2] = temp.pieces[x][y];
					else
						temp.pieces[x+1][y+2] = white_king;
					temp.pieces[x][y] = empty;
					temp.pieces[x+1][y+1] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(y%2 && x != 0 && b.pieces[x][y+1] > white_king_moved && !b.pieces[x-1][y+2])
				{
					//TODO add double takes here
					if (y != 5)
						temp.pieces[x-1][y+2] = temp.pieces[x][y];
					else
						temp.pieces[x+1][y+2] = white_king;
					temp.pieces[x][y] = empty;
					temp.pieces[x][y+1] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && y < 5 && x != 0 && b.pieces[x-1][y+1] > white_king_moved && !b.pieces[x-1][y+2])
				{
					//TODO add double takes here
					temp.pieces[x-1][y+2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x-1][y+1] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && y < 5 && b.pieces[x][y+1] > white_king_moved && !b.pieces[x+1][y+2])
				{
					//TODO add double takes here
					temp.pieces[x+1][y+2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x][y+1] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
			}
			if (b.pieces[x][y] == white_king)
			{
		
				/*White king move backwards(not take) */
				if(y%2 && x != 3 && !b.pieces[x+1][y-1]) 
				{
					temp.pieces[x+1][y-1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(y%2 && !b.pieces[x][y-1])
				{
					temp.pieces[x][y-1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && y>0 && x != 0 && !b.pieces[x-1][y-1])
				{
					temp.pieces[x-1][y-1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && y>0 && !b.pieces[x][y-1])
				{
					temp.pieces[x][y-1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(y%2 && y>1 && x!= 3 && b.pieces[x+1][y-1] > white_king_moved && !b.pieces[x+1][y-2]) 
				{
					//TODO add double takes here
					temp.pieces[x+1][y-2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x+1][y-1] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(y%2 && y>1 && x != 0 && b.pieces[x][y-1] > white_king_moved && !b.pieces[x-1][y-2])
				{
					//TODO add double takes here
					temp.pieces[x-1][y-2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x][y-1] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && y>0 && x != 0 && b.pieces[x-1][y-1] > white_king_moved && !b.pieces[x-1][y-2])
				{
					//TODO add double takes here
					temp.pieces[x-1][y-2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x-1][y-1] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && y>0 && b.pieces[x][y-1] > white_king_moved && !b.pieces[x+1][y-2])
				{
					//TODO add double takes here
					temp.pieces[x+1][y-2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x][y-1] = empty;
					boards[tx+move_idx*exp_rate] = temp;
					move_idx++;
					temp = b;
				}
			}
		}
	}
	else if (tx < 22)
	{
		int move_idx = 0;
		Board b = boards[tx*22];
		Board temp = boards[tx*22];
		for(int x = 0; x < 4; x++)
		for(int y = 0; y < 8; y++)
		{
			if (b.pieces[x][y] == black_reg || b.pieces[x][y] == black_king)
			{
				/*White pieces move (not take) */
				if(y%2 && x != 3 && !b.pieces[x+1][y-1]) 
				{	
					//printf("black at %d,%d move right\n", x, y);
					if (y == 1)
						temp.pieces[x+1][y-1] = black_king;
					else
						temp.pieces[x+1][y-1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(y%2 && !b.pieces[x][y-1])
				{
					//printf("black at %d,%d move left\n", x, y);
					if (y == 1)
						temp.pieces[x+1][y-1] = black_king;
					else
						temp.pieces[x][y-1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && x != 0 && !b.pieces[x-1][y-1])
				{
					//printf("black at %d,%d move left\n", x, y);
					temp.pieces[x-1][y-1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && !b.pieces[x][y-1])
				{
					//printf("black at %d,%d move right\n", x, y);
					temp.pieces[x][y-1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				/*White piece captures a black piece*/
				if(y%2 && y>1 && x!= 3 && b.pieces[x+1][y-1] > 0 && b.pieces[x+1][y-1] <= white_king_moved && !b.pieces[x+1][y-2]) 
				{
					//TODO add double takes here
					if (y != 2)
						temp.pieces[x+1][y-2] = temp.pieces[x][y];
					else
						temp.pieces[x+1][y-2] = white_king;
					temp.pieces[x][y] = empty;
					temp.pieces[x+1][y-1] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(y%2 && y>1 && x != 0 && b.pieces[x][y-1] > 0 && b.pieces[x][y-1] <= white_king_moved && !b.pieces[x-1][y-2])
				{
					//TODO add double takes here
					if (y != 2)
						temp.pieces[x-1][y-2] = temp.pieces[x][y];
					else
						temp.pieces[x+1][y-2] = white_king;
					temp.pieces[x][y] = empty;
					temp.pieces[x][y-1] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && y>2 && x != 0 && b.pieces[x-1][y-1] <= white_king_moved && b.pieces[x-1][y-1] > 0 && !b.pieces[x-1][y-2])
				{
					//TODO add double takes here
					temp.pieces[x-1][y-2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x-1][y-1] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && y>2 && b.pieces[x][y-1] <= white_king_moved && b.pieces[x][y-1]>0 && !b.pieces[x+1][y-2])
				{
					//TODO add double takes here
					temp.pieces[x+1][y-2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x][y-1] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
			}
			if (b.pieces[x][y] == black_king)
			{
		
				/*White king move backwards(not take) */
				if(y%2 && y<7 && x != 3 && !b.pieces[x+1][y+1]) 
				{
					temp.pieces[x+1][y+1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(y%2 && y<7 && !b.pieces[x][y+1])
				{
					temp.pieces[x][y+1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && x != 0 && !b.pieces[x-1][y+1])
				{
					temp.pieces[x-1][y+1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && !b.pieces[x][y+1])
				{
					temp.pieces[x][y+1] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(y%2 && y<6 && x!= 3 && b.pieces[x+1][y+1] <= white_king_moved && b.pieces[x+1][y+1] > 0 && !b.pieces[x+1][y+2]) 
				{
					//TODO add double takes here
					temp.pieces[x+1][y+2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x+1][y+1] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(y%2 && y<6 && x != 0 && b.pieces[x][y+1] <= white_king_moved && b.pieces[x][y+1] > 0 && !b.pieces[x-1][y+2])
				{
					//TODO add double takes here
					temp.pieces[x-1][y+2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x][y+1] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && y<5 && x != 0 && b.pieces[x-1][y+1] <= white_king_moved && b.pieces[x-1][y+1] > 0 && !b.pieces[x-1][y+2])
				{
					//TODO add double takes here
					temp.pieces[x-1][y+2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x-1][y+1] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
				if(!(y%2) && y<5 && b.pieces[x][y+1] <= white_king_moved && b.pieces[x][y+1] > 0 && !b.pieces[x+1][y+2])
				{
					//TODO add double takes here
					temp.pieces[x+1][y+2] = temp.pieces[x][y];
					temp.pieces[x][y] = empty;
					temp.pieces[x][y+1] = empty;
					boards[22*tx+move_idx] = temp;
					move_idx++;
					temp = b;
				}
			}
		}
	}
} 

void printBoard(Board b);
int initBoard(Board *b);
int makeMove(Board *board);
int analyseBoard(Board *board, Turn player);

int main(int argc, char **argv) {
	Board * b = (Board *)malloc(sizeof(Board)*512);
	int moveCount = 1;
	bool drawFlag = false;
	initBoard(b);
	makeMove(b);
	
/*
	while(1)
	{
		printBoard(b[0]);
		makeMove(b);
		for (int i = 0; i < 256; i++)
			for(int a = 0; a < 4; a++)
				for(int j = 0; j < 8; j++)
					if(b[i].pieces[a][j] != 0)
					{
						//printf("B: %d, Loc: (%d, %d), piece: %d\n", i, a, j, b[i].pieces[a][j]);
						printBoard(b[i]);
						break;
					}
		for(int i = 0; i < moveCount; i++)
		{
			int score = analyseBoard(&b[i], white);
			//printf("B: %d Score: %d\n", i, score);
			if(!analyseBoard(&b[i], black)) 
			{
				printBoard(b[i]);
				return 0;
			}
			//printBoard(b[i]);

		}
	}
		*/
}
void printBoard(Board b)
{
	printf("Board: --------------------------------------\n");
	for(int i = 3; i >= 0; i--)
	{
		for(int j = 0; j < 4; j++)
		{
			switch(b.pieces[j][i*2+1])
			{
				case white_reg:
				case white_reg_moved:
					printf("_|w|");
					break;
				case white_king:
				case white_king_moved:
					printf("_|W|");
					break;
				case black_reg:
				case black_reg_moved:
					printf("_|b|");
					break;
				case black_king:
				case black_king_moved:
					printf("_|B|");
					break;
				case empty:
					printf("_|_|");
					break;
				default:
					printf("x|x|");
					break;
			}
		}
		printf("\n");
		for(int j = 0; j < 4; j++)
		{
			switch(b.pieces[j][i*2])
			{
				case white_reg:
				case white_reg_moved:
					printf("w|_|");
					break;
				case white_king:
				case white_king_moved:
					printf("W|_|");
					break;
				case black_reg:
				case black_reg_moved:
					printf("b|_|");
					break;
				case black_king:
				case black_king_moved:
					printf("B|_|");
					break;
				case empty:
					printf("_|_|");
					break;
				default:
					printf("x|x|");
					break;
			}
		}
		printf("\n");
	}
}

int initBoard(Board *board)
{
	if(!board)
	{
		return -1;
	}
	for(int y = 0; y < 3; y++)
	{
		for(int x = 0; x < 4; x++)
		{
			board->pieces[x][y] = white_reg;
			board->pieces[x][y + 5] = black_reg;
		}
	}
	return 0;
}

int makeMove(Board *board)
{
	Board *host_output;
	Board *host_input;
	Board *device_output;
	Board *device_input;

	int inputSize = 1;
	int outputSize = inputSize * 512;
	
	host_input =  board;
	#if USE_GPU

	// cuda malloc
	hipMalloc(&device_output, outputSize * sizeof(Board));
	hipMalloc(&device_input, inputSize * sizeof(Board));
	
	// cuda memcpy
	hipMemcpy(device_input, host_input, inputSize * sizeof(*device_input), hipMemcpyHostToDevice);

	//launch kernel and check errors
	//printf("initializing kernel with grid dim: %d and block dim: %d\n", inputSize, BLOCK_SIZE);
	dim3 dimGrid(inputSize);
	dim3 dimBlock(BLOCK_SIZE);
	expand<<<dimGrid, dimBlock>>>(device_input, device_output, inputSize);
	hipPeekAtLastError();
	hipDeviceSynchronize();

	//set up for second kernel launch
	inputSize = outputSize;
	outputSize = inputSize * 512;
	hipFree(device_input);
	device_input = device_output;
	hipMalloc(&device_output, outputSize * sizeof(Board));
	
	//launch kernel and check errors
	//printf("initializing kernel with grid dim: %d and block dim: %d\n", inputSize, BLOCK_SIZE);
	dim3 dimGrid2(inputSize);
	expand<<<dimGrid2, dimBlock>>>(device_input, device_output,	inputSize);
	hipPeekAtLastError();
	hipDeviceSynchronize();
	
	//print all boards after 2 full turns have been taken
	/*host_output = (Board *) malloc(outputSize*sizeof(*host_output));
	gpuErrChk(cudaMemcpy(host_output, device_output, outputSize * sizeof(*device_input),
				cudaMemcpyDeviceToHost));
	for(int i = 0; i < outputSize; i++)
		if (!boardEqualityHost(&bad_board_host, &host_output[i]))	
		{	
			int a = 0;
			//printBoard(host_output[i]);		
			//printf("Board #: %d", i);
		}
*/
	for(int i = 0; i < 512*512; i++)
	{
		device_input = &device_output[i];
		Board *temp_device_output;
		hipMalloc(&temp_device_output, 512*sizeof(Board));
		dim3 dimGrid(1);
		dim3 dimBlock(512);
		expand<<<dimGrid, dimBlock>>>(device_input, temp_device_output, 1);
		hipPeekAtLastError();
		hipDeviceSynchronize();
		
		device_input = temp_device_output;
		gpuErrChk(hipMalloc(&temp_device_output, 512*512*sizeof(Board)));
		dim3 dimGrid2(512);
		expand<<<dimGrid2, dimBlock>>>(device_input, temp_device_output, 512);
		hipPeekAtLastError();
		hipDeviceSynchronize();

		host_output = (Board *) malloc(512*512*sizeof(*host_output));
		gpuErrChk(hipMemcpy(host_output, temp_device_output, 512*512*sizeof(Board),
					hipMemcpyDeviceToHost));
		gpuErrChk(hipFree(temp_device_output));
		gpuErrChk(hipFree(device_input));
		//for(int i = 0; i < 512*512; i++)
			//if (!boardEqualityHost(&bad_board_host, &host_output[i]))	
			if(i == 512*512-1)
			{	
				int a = 0;
				printBoard(host_output[0]);		
				printf("Board #: %d", 0);
			}	
	
	}



	#endif
	

	return 0;
}

int analyseBoard(Board *board, Turn player)
{
	int score = 0;
	uint8_t pieceMin, pieceMax;
	if(player == white)
	{
		pieceMin = white_reg;
		pieceMax = white_king_moved;
	} else
	{
		pieceMin = black_reg;
		pieceMax = black_reg_moved;
	}

	for(int x = 0; x < 4; x++)
	{
		for(int y = 0; y < 8; y++)
		{
			Piece piece = board->pieces[x][y];
			if(pieceMin <= piece && piece <= pieceMax)
			{
				score++;
			}
		}
	}
	return score;		






}
